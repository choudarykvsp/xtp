#include <algorithm>
#include <string>
#include <votca/xtp/aoshell.h>
#include <votca/xtp/gridbox.h>
#include <thrust/host_vector.h>
#include <votca/xtp/gpu/GPUNumericalIntegration.h>
#include <votca/xtp/gpu/GPUCTypes.cuh>
#include "kernels/IntegrationKernels.cuh"

namespace votca { namespace xtp { namespace gpu {

GPUNumericalIntegration::GPUNumericalIntegration(const AOBasis& aob,
                                                 const std::vector<GridBox>& gbs){
    _device=0;
    CUDA_API_CALL(hipSetDevice(_device), true); // add some code here later to intelligently
                                                 // select a gpu
    
    // Later we need to also dissallow this gpu from begin used
    // by other processes (optionally?)...
    _gpuAOB = GPUAOBasis(aob);
    _gpuGridBox = GPUGridBox(gbs);

    size_t baseNumFuncs = _gpuAOB.sConts.size()*_gpuGridBox.h_gridPoints.x.size();

    //allocate funcvalues
    d_sFuncVals = gpu_vector(1 * baseNumFuncs, 0); 
    d_pFuncVals = gpu_vector(3 * baseNumFuncs, 0); 
    d_dFuncVals = gpu_vector(5 * baseNumFuncs, 0); 
    d_fFuncVals = gpu_vector(7 * baseNumFuncs, 0); 
    d_gFuncVals = gpu_vector(9 * baseNumFuncs, 0);

    _gpuFuncVals.sFuncVals.array = thrust::raw_pointer_cast(&d_sFuncVals[0]);
    _gpuFuncVals.sFuncVals.arraySize = d_sFuncVals.size();

    _gpuFuncVals.pFuncVals.array = thrust::raw_pointer_cast(&d_pFuncVals[0]);
    _gpuFuncVals.pFuncVals.arraySize = d_pFuncVals.size();

    _gpuFuncVals.dFuncVals.array = thrust::raw_pointer_cast(&d_dFuncVals[0]);
    _gpuFuncVals.dFuncVals.arraySize = d_dFuncVals.size();

    _gpuFuncVals.fFuncVals.array = thrust::raw_pointer_cast(&d_fFuncVals[0]);
    _gpuFuncVals.fFuncVals.arraySize = d_fFuncVals.size();

    _gpuFuncVals.gFuncVals.array = thrust::raw_pointer_cast(&d_gFuncVals[0]);
    _gpuFuncVals.gFuncVals.arraySize = d_gFuncVals.size();
    
}

}

}}}
