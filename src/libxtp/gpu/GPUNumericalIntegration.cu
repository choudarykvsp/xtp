#include <algorithm>
#include <string>
#include <votca/xtp/aoshell.h>
#include <votca/xtp/gridbox.h>
#include <thrust/host_vector.h>
#include <votca/xtp/gpu/GPUNumericalIntegration.h>
#include <votca/xtp/gpu/GPUCTypes.cuh>
#include "kernels/IntegrationKernels.cuh"

namespace votca { namespace xtp { namespace gpu {

GPUNumericalIntegration::GPUNumericalIntegration(const AOBasis& aob,
                                                 const std::vector<GridBox>& gbs){
    _device=0;
    CUDA_API_CALL(hipSetDevice(_device), true); // add some code here later to intelligently
                                                 // select a gpu
    
    // Later we need to also dissallow this gpu from begin used
    // by other processes (optionally?)...
    _gpuAOB = GPUAOBasis(aob);
    _gpuGridBox = GPUGridBox(gbs);
}

}}}
