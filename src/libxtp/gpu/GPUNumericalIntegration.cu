#include <algorithm>
#include <string>
#include <votca/xtp/aoshell.h>
#include <votca/xtp/gridbox.h>
#include <thrust/host_vector.h>
#include <votca/xtp/gpu/GPUNumericalIntegration.h>
#include <votca/xtp/gpu/GPUCTypes.cuh>
#include "kernels/IntegrationKernels.cuh"

namespace votca { namespace xtp { namespace gpu {

GPUNumericalIntegration::GPUNumericalIntegration(const AOBasis& aob,
                                                 const std::vector<GridBox>& gbs){
    _device=0;
    CUDA_API_CALL(hipSetDevice(_device), true); // add some code here later to intelligently
                                                 // select a gpu
    
    // Later we need to also dissallow this gpu from begin used
    // by other processes (optionally?)...
    _gpuAOB = GPUAOBasis(aob);
    _gpuGridBox = GPUGridBox(gbs);

    size_t baseNumFuncs = _gpuAOB.sConts.size()*_gpuGridBox.h_gridPoints.x.size();

    //allocate funcvalues
    d_sFuncVals = gpu_vector(1 * baseNumFuncs, 0); 
    d_pFuncVals = gpu_vector(3 * baseNumFuncs, 0); 
    d_dFuncVals = gpu_vector(5 * baseNumFuncs, 0); 
    d_fFuncVals = gpu_vector(7 * baseNumFuncs, 0); 
    d_gFuncVals = gpu_vector(9 * baseNumFuncs, 0);

    _gpuFuncVals.sFuncVals.array = thrust::raw_pointer_cast(&d_sFuncVals[0]);
    _gpuFuncVals.sFuncVals.arraySize = d_sFuncVals.size();

    _gpuFuncVals.pFuncVals.array = thrust::raw_pointer_cast(&d_pFuncVals[0]);
    _gpuFuncVals.pFuncVals.arraySize = d_pFuncVals.size();

    _gpuFuncVals.dFuncVals.array = thrust::raw_pointer_cast(&d_dFuncVals[0]);
    _gpuFuncVals.dFuncVals.arraySize = d_dFuncVals.size();

    _gpuFuncVals.fFuncVals.array = thrust::raw_pointer_cast(&d_fFuncVals[0]);
    _gpuFuncVals.fFuncVals.arraySize = d_fFuncVals.size();

    _gpuFuncVals.gFuncVals.array = thrust::raw_pointer_cast(&d_gFuncVals[0]);
    _gpuFuncVals.gFuncVals.arraySize = d_gFuncVals.size();
    
}

void GPUNumericalIntegration::EvaluateFuncs(){
    // run the kernels baby

    // maybe this should be configurable??
    dim3 threadsperblock(512, 512); // this can change depending on hardware.
                                    // might need to be careful with this...

    // threadsperblock.x is the number of functions that are evaluated
    // simultaneously. threadsperblock.y is the number of grid points
    // at which each function is evaluated.

    // s
    dim3 numblocks(1 + _gpuAOB.sConts.size()/threadsperblock.x,
                   1 + _gpuGridBox.h_gridPoints.x.size()/threadsperblock.y);

    EvalSFuncs<<<numblocks, threadsperblock>>>(_gpuAOB.GetRawGPUArrs(),
                                               _gpuGridBox.GetRawGPUArrs(),
                                               _gpuFuncVals.sFuncVals);

    // p
    numblocks = dim3(1 + _gpuAOB.pConts.size()/threadsperblock.x,
                     1 + _gpuGridBox.h_gridPoints.x.size()/threadsperblock.y);

    EvalPFuncs<<<numblocks, threadsperblock>>>(_gpuAOB.GetRawGPUArrs(),
                                               _gpuGridBox.GetRawGPUArrs(),
                                               _gpuFuncVals.pFuncVals);

    // d
    numblocks = dim3(1 + _gpuAOB.dConts.size()/threadsperblock.x,
                     1 + _gpuGridBox.h_gridPoints.x.size()/threadsperblock.y);

    EvalDFuncs<<<numblocks, threadsperblock>>>(_gpuAOB.GetRawGPUArrs(),
                                               _gpuGridBox.GetRawGPUArrs(),
                                               _gpuFuncVals.dFuncVals);

    // f
    numblocks = dim3(1 + _gpuAOB.fConts.size()/threadsperblock.x,
                     1 + _gpuGridBox.h_gridPoints.x.size()/threadsperblock.y);

    EvalFFuncs<<<numblocks, threadsperblock>>>(_gpuAOB.GetRawGPUArrs(),
                                               _gpuGridBox.GetRawGPUArrs(),
                                               _gpuFuncVals.fFuncVals);


    // g
    numblocks = dim3(1 + _gpuAOB.gConts.size()/threadsperblock.x,
                     1 + _gpuGridBox.h_gridPoints.x.size()/threadsperblock.y);

    EvalGFuncs<<<numblocks, threadsperblock>>>(_gpuAOB.GetRawGPUArrs(),
                                               _gpuGridBox.GetRawGPUArrs(),
                                               _gpuFuncVals.gFuncVals);
}

}}}
